#include "hip/hip_runtime.h"

/* 
    Copyright (C) 2009  Benjamin Vernoux, titanmkd@gmail.com

    This library is free software; you can redistribute it and/or
    modify it under the terms of the GNU Lesser General Public
    License as published by the Free Software Foundation; either
    version 3 of the License, or (at your option) any later version.

    This library is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
    Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with this library; if not, write to the Free Software
    Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA */

#include "cuda_md5_gpu.h"

//
// On-device variable declarations
//

__constant__ uint target[4];		// target hash, if searching for hash matches


void init_md5_target_constants(uint *target_cpu)
{
	if(target_cpu) { hipMemcpyToSymbol(HIP_SYMBOL(target), target_cpu, sizeof(target)); };
}

//
// MD5 routines (straight from Wikipedia's MD5 pseudocode description)
//

//////////////////////////////////////////////////////////////////////////////
/////////////       Ron Rivest's MD5 C Implementation       //////////////////
//////////////////////////////////////////////////////////////////////////////

/*
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z))) 

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }

/*
* Basic MD5 step. Transform buf based on i0 to i3 128bits in registers.
 */
void inline __device__ md5_transform(const uint4 i0,
							  uint &a, uint &b, uint &c, uint &d
							  )
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

/*
#define in0  (i0.x)
#define in1  (i0.y)
#define in2  (i0.z)
#define in3  (i0.w)
#define in4  (i1.x)
#define in5  (i1.y)
#define in6  (i1.z)
#define in7  (i1.w)
#define in8  (i2.x)
#define in9  (i2.y)
#define in10 (i2.z)
#define in11 (i2.w)
#define in12 (i3.x)
#define in13 (i3.y)
#define in14 (i3.z)
#define in15 (i3.w)
*/
#define in0  (i0.x)
#define in1  (i0.y)
#define in2  (i0.z)
#define in3  (0)
#define in4  (0)
#define in5  (0)
#define in6  (0)
#define in7  (0)
#define in8  (0)
#define in9  (0)
#define in10 (0)
#define in11 (0)
#define in12 (0)
#define in13 (0)
#define in14 (i0.w)
#define in15 (0)

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

  /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
  FF ( a, b, c, d, in0,  S11, 3614090360); /* 1 */
  FF ( d, a, b, c, in1,  S12, 3905402710); /* 2 */
  FF ( c, d, a, b, in2,  S13,  606105819); /* 3 */
  FF ( b, c, d, a, in3,  S14, 3250441966); /* 4 */
  FF ( a, b, c, d, in4,  S11, 4118548399); /* 5 */
  FF ( d, a, b, c, in5,  S12, 1200080426); /* 6 */
  FF ( c, d, a, b, in6,  S13, 2821735955); /* 7 */
  FF ( b, c, d, a, in7,  S14, 4249261313); /* 8 */
  FF ( a, b, c, d, in8,  S11, 1770035416); /* 9 */
  FF ( d, a, b, c, in9,  S12, 2336552879); /* 10 */
  FF ( c, d, a, b, in10, S13, 4294925233); /* 11 */
  FF ( b, c, d, a, in11, S14, 2304563134); /* 12 */
  FF ( a, b, c, d, in12, S11, 1804603682); /* 13 */
  FF ( d, a, b, c, in13, S12, 4254626195); /* 14 */
  FF ( c, d, a, b, in14, S13, 2792965006); /* 15 */
  FF ( b, c, d, a, in15, S14, 1236535329); /* 16 */
 
  /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
  GG ( a, b, c, d, in1, S21, 4129170786); /* 17 */
  GG ( d, a, b, c, in6, S22, 3225465664); /* 18 */
  GG ( c, d, a, b, in11, S23,  643717713); /* 19 */
  GG ( b, c, d, a, in0, S24, 3921069994); /* 20 */
  GG ( a, b, c, d, in5, S21, 3593408605); /* 21 */
  GG ( d, a, b, c, in10, S22,   38016083); /* 22 */
  GG ( c, d, a, b, in15, S23, 3634488961); /* 23 */
  GG ( b, c, d, a, in4, S24, 3889429448); /* 24 */
  GG ( a, b, c, d, in9, S21,  568446438); /* 25 */
  GG ( d, a, b, c, in14, S22, 3275163606); /* 26 */
  GG ( c, d, a, b, in3, S23, 4107603335); /* 27 */
  GG ( b, c, d, a, in8, S24, 1163531501); /* 28 */
  GG ( a, b, c, d, in13, S21, 2850285829); /* 29 */
  GG ( d, a, b, c, in2, S22, 4243563512); /* 30 */
  GG ( c, d, a, b, in7, S23, 1735328473); /* 31 */
  GG ( b, c, d, a, in12, S24, 2368359562); /* 32 */

  /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
  HH ( a, b, c, d, in5, S31, 4294588738); /* 33 */
  HH ( d, a, b, c, in8, S32, 2272392833); /* 34 */
  HH ( c, d, a, b, in11, S33, 1839030562); /* 35 */
  HH ( b, c, d, a, in14, S34, 4259657740); /* 36 */
  HH ( a, b, c, d, in1, S31, 2763975236); /* 37 */
  HH ( d, a, b, c, in4, S32, 1272893353); /* 38 */
  HH ( c, d, a, b, in7, S33, 4139469664); /* 39 */
  HH ( b, c, d, a, in10, S34, 3200236656); /* 40 */
  HH ( a, b, c, d, in13, S31,  681279174); /* 41 */
  HH ( d, a, b, c, in0, S32, 3936430074); /* 42 */
  HH ( c, d, a, b, in3, S33, 3572445317); /* 43 */
  HH ( b, c, d, a, in6, S34,   76029189); /* 44 */
  HH ( a, b, c, d, in9, S31, 3654602809); /* 45 */
  HH ( d, a, b, c, in12, S32, 3873151461); /* 46 */
  HH ( c, d, a, b, in15, S33,  530742520); /* 47 */
  HH ( b, c, d, a, in2, S34, 3299628645); /* 48 */

  /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
  II ( a, b, c, d, in0, S41, 4096336452); /* 49 */
  II ( d, a, b, c, in7, S42, 1126891415); /* 50 */
  II ( c, d, a, b, in14, S43, 2878612391); /* 51 */
  II ( b, c, d, a, in5, S44, 4237533241); /* 52 */
  II ( a, b, c, d, in12, S41, 1700485571); /* 53 */
  II ( d, a, b, c, in3, S42, 2399980690); /* 54 */
  II ( c, d, a, b, in10, S43, 4293915773); /* 55 */
  II ( b, c, d, a, in1, S44, 2240044497); /* 56 */
  II ( a, b, c, d, in8, S41, 1873313359); /* 57 */
  II ( d, a, b, c, in15, S42, 4264355552); /* 58 */
  II ( c, d, a, b, in6, S43, 2734768916); /* 59 */
  II ( b, c, d, a, in13, S44, 1309151649); /* 60 */
  II ( a, b, c, d, in4, S41, 4149444226); /* 61 */
  II ( d, a, b, c, in11, S42, 3174756917); /* 62 */
  II ( c, d, a, b, in2, S43,  718787259); /* 63 */
  II ( b, c, d, a, in9, S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

}

// The kernel (this is the entrypoint of GPU code)
// Loads the 64-byte word to be hashed from global to shared memory,
// calls the calculation routine, compares to target and flags if a match is found
__global__ void md5_search_bruteforce(char *gwords, uint *succ)
{

	uint4 i0;
	// compute MD5 hash
	uint a, b, c, d;

	int linidx = (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always

	uint4 *ul4src = (uint4 *)gwords;
	uint index = linidx;
	/* Load using 128bits access */
	i0 = ul4src[index];

	md5_transform(i0, a, b, c, d);
	if(a == target[0] && b == target[1] && c == target[2] && d == target[3])
	{
		uint4 success_res;
		success_res.x = i0.x;
		success_res.y = i0.y;
		success_res.z = i0.z;
		success_res.w = 1;
		*((uint4 *)succ) = success_res;
	}

}

/*
Return 0 if no error else return <> 0 for error
*/
// A helper to export the kernel call to C++ code not compiled with nvcc
extern "C" 
int execute_kernel_md5_search(int blocks_x, int blocks_y, int threads_per_block, int shared_mem_required, uint *gpuHash, char *gpuWords)
{
	dim3 grid;
	hipError_t err;

	grid.x = blocks_x; grid.y = blocks_y;
	
	md5_search_bruteforce<<<grid, threads_per_block, shared_mem_required>>>(gpuWords, gpuHash);	

	err = hipDeviceSynchronize();
	return err;
}
